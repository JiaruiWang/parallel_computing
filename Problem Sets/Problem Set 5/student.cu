#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   ????? The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/


#include "utils.h"
#include <stdio.h>


#include <thrust/device_vector.h>
#include <cstdio>


__global__
void yourHisto1(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               const unsigned int numBins,
               const unsigned int numVals)
{
  //TODO fill in this kernel to calculate the histogram
  //as quickly as possible

  //Although we provide only one kernel skeleton,
  //feel free to use more if it will help you
  //write faster code

  int global_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_index >= numVals) return;

  // __syncthreads();
  atomicAdd(&(histo[vals[global_index]]), 1);

  int sum = 0;
  if (global_index == 0)
  {

    for (int i = 0; i < 1024; ++i)
    {
      printf("histo[%d] = %u\n", i, histo[i]);
      sum += histo[i];
    }
    printf("sum = %d\n", sum);
  }
}


__global__
void yourHisto2(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               const unsigned int numBins,
               const unsigned int numVals)
{

// doesn't work correctly
  int global_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_index >= numVals) return;

  unsigned int l_bins[1024];
  for (int i = 0; i < 1024; ++i)
  {
    l_bins[i] = 0;
  }

  int binIdx = vals[global_index];
  l_bins[binIdx]++;
__syncthreads();
  atomicAdd(&(histo[l_bins[binIdx]]), l_bins[binIdx]);
  // for (int i = 0; i < 1024; ++i)
  // {

  //       atomicAdd(&(histo[i]), l_bins[i]);
  //   __syncthreads();
  // }

  int sum = 0;
  if (global_index == 0)
  {

    for (int i = 0; i < 1024; ++i)
    {
      printf("histo[%d] = %u\n", i, histo[i]);
      sum += histo[i];
    }
    printf("sum = %d\n", sum);
  }
  // atomicAdd(&(histo[threadIdx.x]), s_bins[threadIdx.x]);
}

__global__
void yourHisto3(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               const unsigned int numBins,
               const unsigned int numVals)
{


  int global_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_index* 20>= numVals) return;

  // extern __shared__ unsigned int s_bins[];

  unsigned int l_vals[20];
  for (int i = 0; i < 20; ++i)
  {
    l_vals[i] = vals[global_index * 20 + i];
  }

  for (int i = 0; i < 20; ++i)
  {

    atomicAdd(&(histo[l_vals[i]]), 1);
  }

  // int sum = 0;
  // if (global_index == 0)
  // {

  //   for (int i = 0; i < 1024; ++i)
  //   {
  //     printf("histo[%d] = %u\n", i, histo[i]);
  //     sum += histo[i];
  //   }
  //   printf("sum = %d\n", sum);
  // }
}

__global__
void yourHisto4(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               const unsigned int numBins,
               const unsigned int numVals)
{


  int global_index = blockIdx.x * blockDim.x + threadIdx.x;
  int local_index = threadIdx.x;
  extern __shared__ unsigned int s_bins[];
  s_bins[local_index] = 0;
  if (global_index* 4>= numVals) return;



  unsigned int l_vals[4];
      __syncthreads();

  for (int i = 0; i < 4; ++i)
  {
    l_vals[i] = vals[global_index * 4 + i];
  }
  __syncthreads();

  for (int i = 0; i < 4; ++i)
  {

    atomicAdd(&(s_bins[l_vals[i]]), 1);
  }
  __syncthreads();
  atomicAdd(&(histo[local_index]), s_bins[local_index]);
  
  // __syncthreads();
  // int sum = 0;
  // if (global_index == 0)
  // {

  //   for (int i = 0; i < 1024; ++i)
  //   {
  //     printf("histo[%d] = %u\n", i, histo[i]);
  //     sum += histo[i];
  //   }
  //   printf("sum = %d\n", sum);
  // }
}

__global__
void yourHisto5(const unsigned int* const vals, //INPUT
               unsigned int*  d_bins,      //OUPUT
               const unsigned int numBins,
               const unsigned int numVals)
{


  int global_index = blockIdx.x * blockDim.x + threadIdx.x;
  int local_index = threadIdx.x;
  extern __shared__ unsigned int s_bins[];
  s_bins[local_index] = 0;
  if (global_index* 4>= numVals) return;



  unsigned int l_vals[4];
      __syncthreads();

  for (int i = 0; i < 4; ++i)
  {
    l_vals[i] = vals[global_index * 4 + i];
  }
  __syncthreads();

  for (int i = 0; i < 4; ++i)
  {

    atomicAdd(&(s_bins[l_vals[i]]), 1);
  }
  __syncthreads();
  d_bins[blockIdx.x * 1024 + local_index] = s_bins[local_index];
  // atomicAdd(&(histo[local_index]), s_bins[local_index]);


  
  // __syncthreads();
  // int sum = 0;
  // if (global_index == 0)
  // {

  //   for (int i = 0; i < 1024; ++i)
  //   {
  //     printf("histo[%d] = %u\n", i, histo[i]);
  //     sum += histo[i];
  //   }
  //   printf("sum = %d\n", sum);
  // }
}

__global__
void add_bins(unsigned int* d_bins,
              unsigned int* const d_histo,
              unsigned int blocks)
{
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int sum = 0;
  for (int i = 0; i < blocks; ++i)
  {
    sum += d_bins[i * 1024 + index];
  }
  d_histo[index] = sum;
}



//Udacity HW 4
//Radix Sorting


/* Red Eye Removal
   ===============
   
   For this assignment we are implementing red eye removal.  This is
   accomplished by first creating a score for every pixel that tells us how
   likely it is to be a red eye pixel.  We have already done this for you - you
   are receiving the scores and need to sort them in ascending order so that we
   know which pixels to alter to remove the red eye.

   Note: ascending order == smallest to largest

   Each score is associated with a position, when you sort the scores, you must
   also move the positions accordingly.

   Implementing Parallel Radix Sort with CUDA
   ==========================================

   The basic idea is to construct a histogram on each pass of how many of each
   "digit" there are.   Then we scan this histogram so that we know where to put
   the output of each digit.  For example, the first 1 must come after all the
   0s so we have to know how many 0s there are to be able to start moving 1s
   into the correct position.

   1) Histogram of the number of occurrences of each digit
   2) Exclusive Prefix Sum of Histogram
   3) Determine relative offset of each digit
        For example [0 0 1 1 0 0 1]
                ->  [0 1 0 1 2 3 2]
   4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there

   LSB Radix sort is an out-of-place sort and you will need to ping-pong values
   between the input and output buffers we have provided.  Make sure the final
   sorted results end up in the output buffer!  Hint: You may need to do a copy
   at the end.

 */

__global__ void histogram_kernel(unsigned int* const d_inputVals_t,
                                 unsigned int* const d_inputPos_t,
                                 unsigned int* const d_digits,
                                 unsigned int* const d_histogram,
                                 unsigned int digit,
                                 unsigned int i,
                                 unsigned int numElems)
{
  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index >= numElems) return;

  d_digits[index] = (d_inputVals_t[index] & digit) >> i;
  unsigned int bin = d_digits[index];
  unsigned int step = 1;
  atomicAdd(&(d_histogram[bin]), step);

  // if (index == 0)
  // {
  //   printf("numElems = %d\n", numElems);
  //   printf("d_digits[%d] = %d\n", index, d_digits[index]);
  //   printf("hist : 0 = %d, 1 = %d\n", d_histogram[0], d_histogram[1]);
  // }
}

__global__ void block_exclusive_scan_kernel(unsigned int* d_digits,
                                            unsigned int* d_digits_pos,
                                            unsigned int* d_blelloch_inter,
                                             unsigned int numElems)
{
  extern __shared__ unsigned int sh_mem[];
  int global_index = blockIdx.x * blockDim.x + threadIdx.x;
  int block_index = threadIdx.x;
  if( global_index >= numElems) sh_mem[block_index] = 0;
  else sh_mem[block_index] = d_digits[global_index];
__syncthreads();
//   if (global_index == 1024)
//   {
//     for (int i = 0; i < 1024; ++i)
//     {
//       printf("sh_mem_0[%d] =%u, \n", i, sh_mem[i]);
//     }
//   }
// __syncthreads();
    // Blelloch scan
  

  for (int i = 2; i <= 1024; i *= 2)
  {
    if ((block_index+1)%i == 0)
    {
      int temp = sh_mem[block_index];
      int temp2 = sh_mem[block_index - i/2];
      sh_mem[block_index] = temp + temp2;
      
    }
    __syncthreads();

  }

  if (block_index == 1023)
  {
   sh_mem[1023] = 0;
  }
  __syncthreads();

  
  for (int i = 1024; i >= 2; i = i/2)
  {
    __syncthreads();
    if ((block_index + 1) % i == 0)
    {
      int temp = sh_mem[block_index];
      int temp2 =sh_mem[block_index - i/2];
      sh_mem[block_index - i/2] = temp;
      sh_mem[block_index] = temp + temp2;
    }
    __syncthreads();

  }
  
  if (block_index == 1023)
  {
   d_blelloch_inter[blockIdx.x] = sh_mem[block_index] + d_digits[global_index];
   // printf("blockidx.x =%d\n", blockIdx.x);
  }
  __syncthreads();
  if (global_index < numElems) d_digits_pos[global_index] = sh_mem[block_index];
  __syncthreads();

//   if (global_index == 220000)
//   {
//     for (int i = 0; i < 1024; ++i)
//     {
//       printf("sh_mem_0[%d] =%u, \n", i, sh_mem[i]);
//     }
//   }
// __syncthreads();
//   if (global_index == 1024)
//   {
//     // int sum = 0;
//     // for (int i = 0; i < 216; ++i)
//     // {
//     //   printf("d_blelloch_inter[%d] = %u, \n", i, d_blelloch_inter[i]);
//     //   sum += d_blelloch_inter[i];
//     // }
//     // printf("sum = %d\n", sum);
//     // for (int i = 0; i < numElems; ++i)
//     // {
//     //   printf("%d:%u\n", i, d_digits_pos[i]);
//     // }
//   }
// __syncthreads();
}


__global__ void self_exclusive_scan_kernel(unsigned int* d_blelloch_inter,
                                            unsigned int numElems)
{
  extern __shared__ unsigned int sh_mem[];
  int block_index = blockDim.x * blockIdx.x + threadIdx.x;
  if (block_index >= numElems) sh_mem[block_index] = 0;
  else sh_mem[block_index] = d_blelloch_inter[block_index];
  __syncthreads();

  for (int i = 2; i <= 1024; i *= 2)
  {
    if ((block_index+1)%i == 0)
    {
      int temp = sh_mem[block_index];
      int temp2 = sh_mem[block_index - i/2];
      sh_mem[block_index] = temp + temp2;
      
    }
    __syncthreads();

  }

  if (block_index == 1023)
  {
   sh_mem[1023] = 0;
  }
  __syncthreads();

  
  for (int i = 1024; i >= 2; i = i/2)
  {
    __syncthreads();
    if ((block_index + 1) % i == 0)
    {
      int temp = sh_mem[block_index];
      int temp2 =sh_mem[block_index - i/2];
      sh_mem[block_index - i/2] = temp;
      sh_mem[block_index] = temp + temp2;
    }
    __syncthreads();

  }

  d_blelloch_inter[block_index] = sh_mem[block_index];
  __syncthreads();
  // if (block_index == 0)
  // {
  //   for (int i = 0; i < numElems; ++i)
  //   {
  //     printf("d_blelloch_inter[%d] = %u\n", i, d_blelloch_inter[i]);
  //   }
  // }
}

__global__ void add_inter_kernel(unsigned int* d_blelloch_inter,
                                 unsigned int* d_digits_pos,
                                 unsigned int numElems)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= numElems) return;

  d_digits_pos[index] = d_digits_pos[index] + d_blelloch_inter[blockIdx.x];

  // if (index == 0)
  // {
  //   for (int i = 0; i < numElems; ++i)
  //   {
  //     printf("%d:%u\n", i, d_digits_pos[i]);
  //   }
  // }
}

__global__ void switch_ones_zeros(unsigned int* d_digits,
                                  unsigned int* d_digits_reverse,
                                  unsigned int numElems)
{
  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index >= numElems) return;

  if (d_digits[index] == 1)
  {
    d_digits_reverse[index] = 0;
  }
  if (d_digits[index] == 0)
  {
    d_digits_reverse[index] = 1;
  }
  __syncthreads();
  // if (index == 0)
  // {
  //   unsigned int sum = 0;
  //   for (int i = 0; i < numElems; ++i)
  //   {
  //     printf("%u: %u->%u ", i, d_digits[i], d_digits_reverse[i]);
  //     sum += d_digits_reverse[i];
  //   }
  //   printf("sum = %u\n", sum);
  // }
}

__global__ void add_1_0_pos(unsigned int* d_digits_1_0_pos,
                            unsigned int* d_digits,
                            unsigned int* d_digits_1_pos,
                            unsigned int* d_digits_reverse,
                            unsigned int* d_digits_0_pos,
                            unsigned int numElems)
{
  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index >= numElems) return;

  if (d_digits[index] == 1)
  {
    d_digits_1_0_pos[index] = d_digits_1_pos[index];
  }
  __syncthreads();
  if (d_digits_reverse[index] == 1)
  {
    d_digits_1_0_pos[index] = d_digits_0_pos[index]; 
  }
  __syncthreads();

  // if (index == 1)
  // {
  //   unsigned int sum = 0;
  //   for (int i = 0; i < numElems; ++i)
  //   {
  //     printf("%u:%u ", i, d_digits_1_0_pos[i]);
  //     sum += d_digits_1_0_pos[i];
  //   }
  //   printf("\nsum = %u\n", sum);
  // }
}

__global__ void add_prefix_to_pos(unsigned int* d_digits_ab_pos,
                                  // unsigned int* d_digits_1_0_pos,
                                  unsigned int* d_digits,
                                  // unsigned int* d_digits_1_pos,
                                  // unsigned int* d_digits_reverse,
                                  // unsigned int* d_digits_0_pos,
                                  unsigned int* d_prefix_sum,
                                  unsigned int numElems)
{
  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index >= numElems) return;

  if (d_digits[index] == 1)
  {
    d_digits_ab_pos[index] = d_digits_ab_pos[index] + d_prefix_sum[1];
  }
  __syncthreads();


  // if (index == 0)
  // {
  //   for (int i = 0; i < numElems; ++i)
  //   {
  //     printf("%u:%u ", i, d_digits_ab_pos[i]);
  //   }
  // }
}

__global__ void move_kernel(unsigned int* d_inputVals_t,
                            unsigned int* d_inputPos_t,
                            unsigned int* d_outputVals_t,
                            unsigned int* d_outputPos_t,
                            unsigned int* d_digits_ab_pos,
                            unsigned int numElems)
{
  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index >= numElems) return;
  
  d_outputVals_t[d_digits_ab_pos[index]] = d_inputVals_t[index];
  d_outputPos_t[d_digits_ab_pos[index]] = d_inputPos_t[index];

  // if (index == 0)
  // {
  //   for (unsigned int i = 0; i < numElems; ++i)
  //   {
  //     if (i == 220474)
  //     {
  //       printf("d_inputVals_t[%u] = %u\nd_inputPos_t[%u] = %u\nd_digits_ab_pos[%u] = %u\n", 
  //         i, d_inputVals_t[i], i, d_inputPos_t[i], i, d_digits_ab_pos[i]);
  //     }
  //     if (i == 220479)
  //     {
  //       printf("d_outputVals_t[%u] = %u\nd_outputPos_t[%u] = %u\n", i, d_outputVals_t[i], i, d_outputPos_t[i]);
  //     }
  //   }
  // }
}

void your_sort(unsigned int* const d_inputVals,
               unsigned int* const d_inputPos,
               unsigned int* const d_outputVals,
               unsigned int* const d_outputPos,
               unsigned int numElems)
{ 
  //TODO
  //PUT YOUR SORT HERE
  int threads = 1024;
  int blocks = ceil(1.0 * numElems/ threads);
  
  unsigned int* d_inputVals_t;
  checkCudaErrors(hipMalloc(&d_inputVals_t, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMemcpy(d_inputVals_t, d_inputVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
  unsigned int* d_inputPos_t;
  checkCudaErrors(hipMalloc(&d_inputPos_t, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMemcpy(d_inputPos_t, d_inputPos, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
  unsigned int* d_outputVals_t;
  checkCudaErrors(hipMalloc(&d_outputVals_t, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMemcpy(d_outputVals_t, d_outputVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
  unsigned int* d_outputPos_t;
  checkCudaErrors(hipMalloc(&d_outputPos_t, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMemcpy(d_outputPos_t, d_outputPos, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));

  unsigned int* d_digits;
  checkCudaErrors(hipMalloc(&d_digits, sizeof(unsigned int) * numElems));
  unsigned int* d_histogram;
  checkCudaErrors(hipMalloc(&d_histogram, sizeof(unsigned int) * 2));
  
  unsigned int* d_prefix_sum;
  checkCudaErrors(hipMalloc(&d_prefix_sum, sizeof(unsigned int) * 2));
  
  unsigned int* d_digits_1_pos;
  checkCudaErrors(hipMalloc(&d_digits_1_pos, sizeof(unsigned int) * numElems));
  unsigned int* d_digits_0_pos;
  checkCudaErrors(hipMalloc(&d_digits_0_pos, sizeof(unsigned int) * numElems));
  unsigned int* d_digits_reverse;
  checkCudaErrors(hipMalloc(&d_digits_reverse, sizeof(unsigned int) * numElems));
  unsigned int* d_digits_1_0_pos;
  checkCudaErrors(hipMalloc(&d_digits_1_0_pos, sizeof(unsigned int) * numElems));
  unsigned int* d_digits_ab_pos;
  checkCudaErrors(hipMalloc(&d_digits_ab_pos, sizeof(unsigned int) * numElems));
  unsigned int* d_blelloch_inter;
  checkCudaErrors(hipMalloc(&d_blelloch_inter, sizeof(unsigned int) * blocks));
  unsigned int* d_blelloch_2_inter;
  checkCudaErrors(hipMalloc(&d_blelloch_2_inter, sizeof(unsigned int) * 10));
  unsigned int* d_inter_pos;
  checkCudaErrors(hipMalloc(&d_inter_pos, sizeof(unsigned int) * 10000));

  unsigned int one = 1;
  unsigned int thirtyTwo = (unsigned int)sizeof(unsigned int) * 8;
  for (unsigned int i = 0; i < 10; ++i)
  {
    // 1)
    // printf("%u, one << i = %u, sizeof(unsigned int) * 8 = %u\n", i, one << i,(unsigned int)sizeof(unsigned int) * 8);
    checkCudaErrors(hipMemset(d_histogram, 0, sizeof(unsigned int) * 2));
    checkCudaErrors(hipMemset(d_prefix_sum, 0, sizeof(unsigned int) * 2));
    checkCudaErrors(hipMemset(d_digits_1_pos, 0, sizeof(unsigned int) * numElems));
    
    histogram_kernel<<<blocks, threads>>>(d_inputVals_t, d_inputPos_t, d_digits, d_histogram, one << i, i, numElems);
    // printf("1\n");
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    // 2)
    // printf("2\n");
    checkCudaErrors(hipMemcpy(&(d_prefix_sum[1]), &(d_histogram[0]), sizeof(unsigned int), hipMemcpyDeviceToDevice));

    // unsigned int* h_prefix_sum = (unsigned int*)malloc(sizeof(unsigned int) * 2);
    // checkCudaErrors(hipMemcpy(h_prefix_sum, d_prefix_sum, sizeof(unsigned int) * 2, hipMemcpyDeviceToHost));
    // printf("d_prefix_sum[0]= %d, d_prefix_sum[1] = %d\n", h_prefix_sum[0], h_prefix_sum[1]);
    // hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    // 3)
    checkCudaErrors(hipMemcpy(d_digits_1_pos, d_digits, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemset(d_blelloch_inter, 0, sizeof(unsigned int) * blocks));
    checkCudaErrors(hipMemset(d_blelloch_2_inter, 0, sizeof(unsigned int) * 10));
    checkCudaErrors(hipMemset(d_inter_pos, 0, sizeof(unsigned int) * 10000));
    block_exclusive_scan_kernel<<<blocks, threads, sizeof(unsigned int) * 1024>>>(d_digits, d_digits_1_pos, d_blelloch_inter, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    block_exclusive_scan_kernel<<<10, 1000, sizeof(unsigned int) * 1024>>>(d_blelloch_inter, d_inter_pos, d_blelloch_2_inter, 10000);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    self_exclusive_scan_kernel<<<1, threads, sizeof(unsigned int) * 1024>>>(d_blelloch_2_inter, 10);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    add_inter_kernel<<<10, 1000>>>(d_blelloch_2_inter, d_inter_pos, 10000);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    add_inter_kernel<<<blocks, threads>>>(d_inter_pos, d_digits_1_pos, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    // unsigned int* h_blelloch_inter = (unsigned int*)malloc(sizeof(unsigned int) * blocks);
    // checkCudaErrors(hipMemcpy(h_blelloch_inter, d_blelloch_inter, sizeof(unsigned int) * blocks, hipMemcpyDeviceToHost));
    // for (int i = 0; i < blocks; ++i)
    // {
    //   printf("d_blelloch_inter[%d]= %d,", i, d_blelloch_inter[i]);
    // }
   
    // hipDeviceSynchronize(); checkCudaErrors(hipGetLastError()); 

    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());


    switch_ones_zeros<<<blocks, threads>>>(d_digits, d_digits_reverse, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    // exclusive_scan_kernel<<<blocks, threads>>>(d_digits_reverse, d_digits_0_pos, numElems);
    checkCudaErrors(hipMemset(d_blelloch_inter, 0, sizeof(unsigned int) * blocks));
    checkCudaErrors(hipMemset(d_blelloch_2_inter, 0, sizeof(unsigned int) * 10));
    checkCudaErrors(hipMemset(d_inter_pos, 0, sizeof(unsigned int) * 10000));    

    block_exclusive_scan_kernel<<<blocks, threads, sizeof(unsigned int) * 1024>>>(d_digits_reverse, d_digits_0_pos, d_blelloch_inter, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    block_exclusive_scan_kernel<<<10, 1000, sizeof(unsigned int) * 1024>>>(d_blelloch_inter, d_inter_pos, d_blelloch_2_inter, 10000);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    self_exclusive_scan_kernel<<<1, threads, sizeof(unsigned int) * 1024>>>(d_blelloch_2_inter, 10);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    add_inter_kernel<<<10, 1000>>>(d_blelloch_2_inter, d_inter_pos, 10000);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    add_inter_kernel<<<blocks, threads>>>(d_inter_pos, d_digits_0_pos, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());


    add_1_0_pos<<<blocks, threads>>>(d_digits_1_0_pos, d_digits, d_digits_1_pos, d_digits_reverse, d_digits_0_pos, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());


    // 4)

    checkCudaErrors(hipMemcpy(d_digits_ab_pos, d_digits_1_0_pos, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
    add_prefix_to_pos<<<blocks, threads>>>(d_digits_ab_pos, d_digits, d_prefix_sum, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    move_kernel<<<blocks, threads>>>(d_inputVals_t, d_inputPos_t, d_outputVals_t, d_outputPos_t, d_digits_ab_pos, numElems);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    std::swap(d_inputVals_t, d_outputVals_t);
    std::swap(d_inputPos_t, d_outputPos_t);
  }
  
  checkCudaErrors(hipMemcpy(d_outputVals, d_inputVals_t, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMemcpy(d_outputPos, d_inputPos_t, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
  checkCudaErrors(hipFree(d_blelloch_inter));
  checkCudaErrors(hipFree(d_inputVals_t));
  checkCudaErrors(hipFree(d_inputPos_t));
  checkCudaErrors(hipFree(d_outputVals_t));
  checkCudaErrors(hipFree(d_outputPos_t));
  checkCudaErrors(hipFree(d_digits));
  checkCudaErrors(hipFree(d_histogram));
  checkCudaErrors(hipFree(d_prefix_sum));
  checkCudaErrors(hipFree(d_digits_1_pos));
  checkCudaErrors(hipFree(d_digits_0_pos));
  checkCudaErrors(hipFree(d_digits_reverse));
  checkCudaErrors(hipFree(d_digits_1_0_pos));
  checkCudaErrors(hipFree(d_digits_ab_pos));
}



void computeHistogram(const unsigned int* const d_vals, //INPUT
                      unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  //TODO Launch the yourHisto kernel

  //if you want to use/launch more than one kernel,
  //feel free
  unsigned int threads = 1024;
  const unsigned int valsPerTh = 4;
  unsigned int blocks = numElems/threads/valsPerTh;

  // yourHisto1<<<blocks, threads>>>(d_vals, d_histo, numBins, numElems);
  // yourHisto2<<<blocks, threads>>>(d_vals, d_histo, numBins, numElems);
  // yourHisto3<<<blocks, threads>>>(d_vals, d_histo, numBins, numElems);
  // yourHisto4<<<blocks, threads, sizeof(unsigned int) * 1024>>>(d_vals, d_histo, numBins, numElems);
  
  // unsigned int* d_bins;
  // checkCudaErrors(hipMalloc(&d_bins, sizeof(unsigned int) * blocks * 1024));
  // checkCudaErrors(hipMemset(d_bins, 0, sizeof(unsigned int) * blocks * 1024));
  // yourHisto5<<<blocks, threads, sizeof(unsigned int) * 1024>>>(d_vals, d_bins, numBins, numElems);
  // hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  // add_bins<<<1, 1024>>>(d_bins, d_histo, blocks);
  unsigned int* d_outputVals;
  unsigned int* d_inputVals;
  unsigned int* d_inputPos;
  unsigned int* d_outputPos;
  checkCudaErrors(hipMalloc(&d_inputVals, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMemcpy(d_inputVals, d_vals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToDevice));
  checkCudaErrors(hipMalloc(&d_outputVals, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMemset(d_outputVals, 0, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMalloc(&d_inputPos, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMemset(d_inputPos, 0, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMalloc(&d_outputPos, sizeof(unsigned int) * numElems));
  checkCudaErrors(hipMemset(d_outputPos, 0, sizeof(unsigned int) * numElems));

  your_sort(d_inputVals, d_inputPos, d_outputVals, d_outputPos, numElems);

  unsigned int* h_outputVals = (unsigned int*)malloc(sizeof(unsigned int) * numElems);
  checkCudaErrors(hipMemcpy(h_outputVals, d_outputVals, sizeof(unsigned int) * numElems, hipMemcpyDeviceToHost));
  unsigned int* h_outputPos = (unsigned int*)malloc(sizeof(unsigned int) * numElems);
  checkCudaErrors(hipMemcpy(h_outputPos, d_outputPos, sizeof(unsigned int) * numElems, hipMemcpyDeviceToHost));
  for (int i = 0; i < numElems; ++i)
  {
    printf("%d : pos[%u] : v(%u)\n", i, h_outputPos[i], h_outputVals[i]);
  }
}
